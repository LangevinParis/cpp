#include <cstdio>
#include <hip/hip_runtime.h>  //Useless?

#include <iostream>
using namespace std;

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n"); // Does not print...

}

int main() {

    cuda_hello<<<1,1>>>(); 
    
    cout << "Hello, World from CPU! This can be compiled by nvcc\n";

    return 0;
}
